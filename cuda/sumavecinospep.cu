#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 256
#define T 32
#define V 5


__global__ void sumavecinos(float *a, int n, float *b, int v)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x; // id global
    int size = n/64; // tamaño de la memoria compartida
    __shared__ float temp[size];
	for(int i=tid-v;i<=tid+v;i++)
	{
		if(i>=0 && i < n) // caso en que i esta dentro de los extremos
            temp[threadIdx.x] += a[i];
        else if(i<0) //caso que el i sea negativo
            temp[threadIdx.x] += a[n+(i%n)];
        else // caso en que i sea mayor que el tamaño del arreglo
            temp[threadIdx.x] += a[(i%n)-1];
    }
    __syncthreads(); // se sincronizan las hebras
    if (threadIdx.x == 0) // solamente la primera hebra copia a memoria global
		for (int i=0; i < size; i++)
            b[tid+i] = temp[i];
}


int main() {

	//MEMORIA HOST
	float *h_a = (float *) malloc(N*sizeof(float));
	float *h_b = (float *) malloc(N*sizeof(float));
	// SE INICIALIZA
	for (int i=0; i < N; i++)
		h_a[i] = i;

	//MEMORIA A DEVICE
	float *d_a, *d_b;
	hipMalloc((void **) &d_a, N*sizeof(float));
	hipMalloc((void **) &d_b, N*sizeof(float));
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);

	//LLAMADO A KERNEL
	sumavecinos<<<N/T, T>>>(d_a, N, d_b, V);

	//RESULADO DESDE DEVICE A HOST
	hipMemcpy(h_b, d_b, sizeof(float)*N, hipMemcpyDeviceToHost);

	for (int i=0; i < N; i++)
		printf("%f\n", h_b[i]);

	hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

	exit(0);
}

