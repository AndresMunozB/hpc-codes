
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void suma(double* a, double* b, double* c, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n)
        c[tid] = a[tid] + b[tid];
}

__host__ int main(int argc,char* argv[])
{
    // Tamaño de los vectores
    int n = 1024*1024;

    double *h_a = (double*)malloc(n*sizeof(double));
    double *h_b = (double*)malloc(n*sizeof(double));
    double *h_c = (double*)malloc(n*sizeof(double));

    double *d_a;
    double *d_b;
    double *d_c;

    hipMalloc(&d_a, n*sizeof(double));
    hipMalloc(&d_b, n*sizeof(double));
    hipMalloc(&d_c, n*sizeof(double));

    for(int i = 0; i < n; i++ ) {
        h_a[i] = 1.0;
        h_b[i] = 2.0;
    }

    hipMemcpy( d_a, h_a, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, n*sizeof(double), hipMemcpyHostToDevice);

    int blockSize, gridSize;
 
    blockSize = 128; //threads
    gridSize = (int)ceil((float)n/blockSize); //blocks
    printf("%i\n", gridSize);
    
    suma<<<gridSize, blockSize>>>(d_a, d_b, d_c,n);
    hipMemcpy( h_c, d_c, n*sizeof(double), hipMemcpyDeviceToHost );

    for(int i=0;i<n;i++)
        printf("%f ", h_c[i]);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}