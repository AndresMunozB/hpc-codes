
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define T 32
#define V 5


__global__ void sumavecinos(float *a, int n, float *b, int v)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int sum=0;
	for(int i=tid-v;i<=tid+v;i++)
	{
		printf("%d\n",i);
		if(i>=0 && i < N)
			sum= sum + a[i];
	}
	b[tid] = sum;	
}


int main() {

	//MEMORIA HOST
	float *h_a = (float *) malloc(N*sizeof(float));
	float *h_b = (float *) malloc(N*sizeof(float));
	// SE INICIALIZA
	for (int i=0; i < N; i++)
		h_a[i] = i;

	//MEMORIA A DEVICE
	float *d_a, *d_b;
	hipMalloc((void **) &d_a, N*sizeof(float));
	hipMalloc((void **) &d_b, N*sizeof(float));
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);

	//LLAMADO A KERNEL
	sumavecinos<<<N/T, T>>>(d_a, N, d_b, V);

	//RESULADO DESDE DEVICE A HOST
	hipMemcpy(h_b, d_b, sizeof(float), hipMemcpyDeviceToHost);

	/*for (int i=0; i < N; i++)
		printf("%f\n", h_b[i]);*/

	hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

	exit(0);
}

