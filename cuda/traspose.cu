
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void traspose(double *A, int N)
{
    int i, j;
	i = blockDim.x*blockIdx.x + threadIdx.x;  // global index x (horizontal)
	j = blockDim.y*blockIdx.y + threadIdx.y;  // global index y (vertical)

	float temp;
	if (i > j) {
		temp = A[j*N + i];
		A[j*N + i] = A[i*N + j];
		A[i*N + j] = temp;
	}
}

__host__ void matrix_print(double *m, int n)
{
    for (int i=0;i<n;i++){
        for(int j=0;j<n;j++)
            printf("%f ", m[i*n + j]);
        printf("\n");
    }  
}

__host__ int main(int argc,char* argv[])
{
    // Tamaño de los vectores
    int n = 8;

    double *h_a = (double*)malloc(n*n*sizeof(double));
    double *d_a;
    hipMalloc(&d_a, n*n*sizeof(double));

    for(int i = 0; i < n*n; i++ )
        h_a[i] = i;
    matrix_print(h_a,n);
    printf("\n");
    hipMemcpy( d_a, h_a, n*n*sizeof(double), hipMemcpyHostToDevice);

    int blockSize, gridSize;
 
    blockSize = 4; //threads
    gridSize = (int)ceil((float)n/blockSize); //blocks
    //printf("%i\n", gridSize);
    
    traspose<<<gridSize, blockSize>>>(d_a, n);

    hipMemcpy( h_a, d_a, n*n*sizeof(double), hipMemcpyDeviceToHost );

    matrix_print(h_a,n);
    
    hipFree(d_a);
    free(h_a);
    return 0;
}

