
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

__host__ int main() {
    print_kernel<<<10, 10>>>();
    hipDeviceSynchronize();
}