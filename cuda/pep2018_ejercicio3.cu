#include "hip/hip_runtime.h"
__global__ void sumreduction(int *A, int N, int *sum ) {

    // Declare memoria compartida para el bloque
    __shared__ temp[blockDim.x];
    int NCopy = N; 

    int tid = blockDim.x*blockIDx.x + threadIdx.x; // ID global de la hebra
    
    // Cargar bloque de memoria compartida
    //Cada hebra carga su posición
    temp[threadIdx.x] = A[tid];

    // Sincronizar a que todas hayan terminado
    __syncthreads();

    // Reduccion iterativa dentro del bloque
    while(N > 1){
        if(threadIdx.x < N/2){
            temp[threadIdx.x] = temp[threadIdx.x] + temp[threadIdx.x + N/2)
        }
        __syncthreads();
        N = N/2;
    }

    for(int i = N ; i > 1; i/=2 ){
        if(threadIdx.x < i/2){
            temp[threadIdx.x] = temp[threadIdx.x] + temp[threadIdx.x + i/2)
        }
        __syncthreads();
    }


    // Reduccion total a memoria global sum
    if(threadIdx.x == 0){
        atomicAdd(&sum,temp[0]);
    }
}
    
    __host__ main(){
        
    }